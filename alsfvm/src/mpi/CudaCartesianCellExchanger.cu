#include "hip/hip_runtime.h"
/* Copyright (c) 2018 ETH Zurich, Kjetil Olsen Lye
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "alsfvm/mpi/CudaCartesianCellExchanger.hpp"
#include "alsutils/mpi/safe_call.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"
#include "alsfvm/mpi/cartesian/rank_component.hpp"
#include "alsfvm/mpi/cartesian/opposite_corner.hpp"
#include "alsfvm/mpi/cartesian/opposite_side.hpp"

namespace alsfvm {
namespace mpi {

namespace {

__global__ void extractSideDevice(memory::View<real> output,
    memory::View<const real> input,
    ivec3 start, ivec3 end) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const int nx = end.x - start.x;
    const int ny = end.y - start.y;
    const int nz = end.z - start.z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int inputX = x + start.x;
    const int inputY = y + start.y;
    const int inputZ = z + start.z;

    output.at(x, y, z) = input.at(inputX, inputY, inputZ);

}


__global__ void insertSideDevice(memory::View<real> output,
    memory::View< real> input,
    ivec3 start, ivec3 end) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const int nx = end.x - start.x;
    const int ny = end.y - start.y;
    const int nz = end.z - start.z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int outputX = x + start.x;
    const int outputY = y + start.y;
    const int outputZ = z + start.z;

    output.at(outputX, outputY, outputZ) = input.at(x, y, z);

}

}
CudaCartesianCellExchanger::CudaCartesianCellExchanger(ConfigurationPtr&
    configuration, const ivec6& neighbours, const std::array<int, 8>& cornerNeighbours)
    : configuration(configuration), neighbours(neighbours), cornerNeighbours(cornerNeighbours) {


}

RequestContainer CudaCartesianCellExchanger::exchangeCells(
    volume::Volume& outputVolume,
    const volume::Volume& inputVolume) {

    const int dimensions = inputVolume.getDimensions();

    if (buffers.size() == 0) {
        makeBuffers(inputVolume, outputVolume);
        makeStreams(inputVolume);

        receiveRequests.resize(inputVolume.getNumberOfVariables());
        sendRequests.resize(inputVolume.getNumberOfVariables());

        receiveRequestsCorners.resize(inputVolume.getNumberOfVariables());
        sendRequestsCorners.resize(inputVolume.getNumberOfVariables());

        for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
            receiveRequests[var].resize(2 * dimensions);
            sendRequests[var].resize(2 * dimensions);

            receiveRequestsCorners[var].resize(numberOfCorners);
            sendRequestsCorners[var].resize(numberOfCorners);
        }
    }

    exchangeSides(outputVolume, inputVolume);
    exchangeCorners(outputVolume, inputVolume);

    CUDA_SAFE_CALL(hipDeviceSynchronize());



    RequestContainer emptyContainer;
    return emptyContainer;
}


void CudaCartesianCellExchanger::exchangeSides(volume::Volume& outputVolume,
                                                const volume::Volume& inputVolume) {




        extractSides(inputVolume);

        RequestContainer container;

        const auto dimensions = outputVolume.getDimensions();

        for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {

            for (int side = 0; side < 2 * dimensions; ++side) {
                if (hasSide(side)) {
                    CUDA_SAFE_CALL(hipStreamSynchronize(memoryStreams[var][side]));
                    sendRequests[var][side] = (Request::isend(cpuBuffersSend[var][side],
                                cpuBuffersSend[var][side].size(),
                                MPI_DOUBLE, neighbours[side],
                                var * 6 + side,
                                *configuration));
                }

                if (hasSide(cartesian::oppositeSide(side))) {
                    receiveRequests[var][cartesian::oppositeSide(side)] = Request::ireceive(
                            cpuBuffersReceive[var][cartesian::oppositeSide(side)],
                            cpuBuffersReceive[var][cartesian::oppositeSide(side)].size(),
                            MPI_DOUBLE, neighbours[cartesian::oppositeSide(side)],
                            var * 6 + side,
                            *configuration);
                }
            }
        }


        insertSides(outputVolume);

}


void CudaCartesianCellExchanger::exchangeCorners(volume::Volume& outputVolume,
                                                const volume::Volume& inputVolume) {




        extractCorners(inputVolume);


        const auto numberOfVariables = inputVolume.getNumberOfVariables();

        RequestContainer container;

        const auto dimensions = outputVolume.getDimensions();

        for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {

            for (int corner = 0; corner < numberOfCorners; ++corner) {
                if (hasCorner(corner)) {
                    CUDA_SAFE_CALL(hipStreamSynchronize(memoryStreamsCorners[var][corner]));
                    sendRequestsCorners[var][corner] = (Request::isend(cpuBuffersSend[var][corner],
                                cpuBuffersSend[var][corner].size(),
                                MPI_DOUBLE, neighbours[corner],
                                6*numberOfVariables + var*numberOfCorners + corner,
                                *configuration));
                }

                if (hasCorner(cartesian::oppositeCorner(dimensions, corner))) {
                    receiveRequestsCorners[var][cartesian::oppositeCorner(dimensions, corner)] = Request::ireceive(
                            cpuBuffersReceive[var][cartesian::oppositeCorner(dimensions, corner)],
                            cpuBuffersReceive[var][cartesian::oppositeCorner(dimensions, corner)].size(),
                            MPI_DOUBLE, neighbours[cartesian::oppositeCorner(dimensions, corner)],
                            6*numberOfVariables + var*numberOfCorners + corner,
                            *configuration);
                }
            }
        }


        insertSides(outputVolume);

}

int CudaCartesianCellExchanger::getNumberOfActiveSides() const {
    int activeSides = 0;

    for (int i = 0; i < 6; ++i) {
        if (hasSide(i)) {
            activeSides++;
        }
    }

    return activeSides;
}

real CudaCartesianCellExchanger::max(real number) {
    real max;
    MPI_SAFE_CALL(MPI_Allreduce(&number, &max, 1, MPI_DOUBLE, MPI_MAX,
            configuration->getCommunicator()));
    return max;
}
bool CudaCartesianCellExchanger::hasSide(int side) const {
    return neighbours[side] != -1;
}

void CudaCartesianCellExchanger::extractSide(const ivec3& start,
    const ivec3& end,
    int side,
    const volume::Volume& inputVolume) {
    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        extractMemory(start, end, *inputVolume.getScalarMemoryArea(var),
                      memoryStreams[side][var],
                      cpuBuffersSend[side][var],
                      *buffers[side][var]
                      );
    }

}

void CudaCartesianCellExchanger::extractMemory(const ivec3& start, const ivec3& end,
        const memory::Memory<real>& inputMemory,
        hipStream_t stream,
                                             thrust::host_vector<real>& cpuBuffer,
                                             memory::Memory<real>& buffer) {


        const auto diff = end - start;
        const int size = diff.x * diff.y * diff.z;

        if (size <= 0) {
            return;
        }

        const int numberOfThreads = 512;

        extractSideDevice <<< (size + numberOfThreads - 1) / numberOfThreads,
                          numberOfThreads,
                          0, stream >>> (buffer.getView(),
                              inputMemory.getView(),
                              start,
                              end);


        CUDA_SAFE_CALL(hipMemcpyAsync(cpuBuffer.data(),
                buffer.getPointer(),
                buffer.getSize()*sizeof(real),
                hipMemcpyDeviceToHost,
                stream));



}

ivec6 CudaCartesianCellExchanger::getNeighbours() const {
    return neighbours;
}
void CudaCartesianCellExchanger::extractSides(const volume::Volume&
    inputVolume) {
    const int nx = inputVolume.getTotalNumberOfXCells();
    const int ny = inputVolume.getTotalNumberOfYCells();
    const int nz = inputVolume.getTotalNumberOfZCells();

    const int ngx = inputVolume.getNumberOfXGhostCells();
    const int ngy = inputVolume.getNumberOfYGhostCells();
    const int ngz = inputVolume.getNumberOfZGhostCells();

    const int dimensions = inputVolume.getDimensions();

    if (hasSide(0)) {
        extractSide({ngx, 0, 0}, {2 * ngx, ny, nz}, 0, inputVolume);
    }

    if (hasSide(1)) {

        extractSide({nx - 2 * ngx, 0, 0}, {nx - ngx, ny, nz}, 1, inputVolume);
    }

    if (dimensions > 1) {

        if (hasSide(2)) {

            extractSide({0, ngy, 0}, {nx, 2 * ngy, nz}, 2, inputVolume);
        }

        if (hasSide(3)) {

            extractSide({0, ny - 2 * ngy, 0}, {nx, ny - ngy, nz}, 3, inputVolume);
        }

        if (dimensions > 2) {
            if (hasSide(4)) {

                extractSide({0, 0, ngz}, {nx, ny, 2 * ngz}, 4, inputVolume);
            }

            if (hasSide(5)) {

                extractSide({0, 0, nz - 2 * ngz}, {nx, ny, nz - ngz}, 5, inputVolume);
            }
        }

    }

}

void CudaCartesianCellExchanger::extractCorners(const volume::Volume& inputVolume) {

    const auto size = inputVolume.getInnerSize();
    const auto gc = inputVolume.getNumberOfGhostCells();

    for (int corner = 0; corner < numberOfCorners; ++corner) {
        if (hasCorner(corner)) {

            const auto position = cartesian::getCoordinates(corner, 2);

            ivec3 start = {0,0,0};
            for (int i = 0; i < 3; ++i) {
                if (position[i] > 0) {
                    start[i] = size[i] + gc[i];
                }
            }

            const auto end = start + gc;
            for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
                extractMemory(start, end, *inputVolume.getScalarMemoryArea(var),
                              memoryStreamsCorners[corner][var],
                              cpuBuffersSendCorners[corner][var],
                              *buffersCorners[corner][var]
                              );
            }
        }
    }
}

bool CudaCartesianCellExchanger::hasCorner(int corner) const {
    return cornerNeighbours[corner] > -1;
}

void CudaCartesianCellExchanger::insertSide(const ivec3& start,
    const ivec3& end,
    int side,
    volume::Volume& outputVolume) {
    for (int var  = 0; var < outputVolume.getNumberOfVariables(); ++var) {


        const auto diff = end - start;
        const int size = diff.x * diff.y * diff.z;

        if (size == 0) {
            return;
        }


        //sendRequests[var][side]->wait();
        receiveRequests[var][side]->wait();

        insertMemory(start, end, *outputVolume.getScalarMemoryArea(var),
                     memoryStreams[var][side],
                     cpuBuffersReceive[var][side],
                     *buffers[var][side]);

    }
}

void CudaCartesianCellExchanger::insertMemory(const ivec3& start,
    const ivec3& end,
    memory::Memory<real>& outputMemory,
    hipStream_t stream,
    thrust::host_vector<real>& cpuBuffer,
    memory::Memory<real>& buffer) {



        const auto diff = end - start;
        const int size = diff.x * diff.y * diff.z;

        if (size == 0) {
            return;
        }




        CUDA_SAFE_CALL(hipMemcpyAsync(buffer.getPointer(),
                cpuBuffer.data(),
                buffer.getSize()*sizeof(real),
                hipMemcpyHostToDevice,
                stream));

        const int numberOfThreads = 512;
        insertSideDevice <<< (size + numberOfThreads - 1) / numberOfThreads,
                         numberOfThreads,
                         0, stream >>> (
                             outputMemory.getView(),
                             buffer.getView(),
                             start,
                             end);


}


void CudaCartesianCellExchanger::insertCorners(volume::Volume& outputVolume) {
    const auto size = outputVolume.getInnerSize();
    const auto gc = outputVolume.getNumberOfGhostCells();

    for (int corner = 0; corner < numberOfCorners; ++corner) {
        if (hasCorner(corner)) {

            const auto position = cartesian::getCoordinates(corner, 2);

            ivec3 start = {0,0,0};
            for (int i = 0; i < 3; ++i) {
                if (position[i] > 0) {
                    start[i] = size[i] + gc[i];
                }
            }

            const auto end = start + gc;
            for (int var = 0; var < outputVolume.getNumberOfVariables(); ++var) {

                receiveRequestsCorners[var][corner]->wait();
                insertMemory(start, end, *outputVolume.getScalarMemoryArea(var),
                              memoryStreamsCorners[corner][var],
                              cpuBuffersReceiveCorners[corner][var],
                              *buffersCorners[corner][var]
                              );
            }
        }
    }
}



void CudaCartesianCellExchanger::insertSides( volume::Volume& outputVolume) {
    const int nx = outputVolume.getTotalNumberOfXCells();
    const int ny = outputVolume.getTotalNumberOfYCells();
    const int nz = outputVolume.getTotalNumberOfZCells();

    const int ngx = outputVolume.getNumberOfXGhostCells();
    const int ngy = outputVolume.getNumberOfYGhostCells();
    const int ngz = outputVolume.getNumberOfZGhostCells();


    const int dimensions = outputVolume.getDimensions();

    if (hasSide(0)) {
        insertSide({0, 0, 0}, {ngx, ny, nz}, 0, outputVolume);
    }

    if (hasSide(1)) {
        insertSide({nx - ngx, 0, 0}, {nx, ny, nz}, 1, outputVolume);
    }

    if (dimensions > 1) {

        if (hasSide(2)) {
            insertSide({0, 0, 0}, {nx, ngy, nz}, 2, outputVolume);
        }

        if (hasSide(3)) {
            insertSide({0, ny - ngy, 0}, {nx, ny, nz}, 3, outputVolume);
        }

        if (dimensions > 2 ) {
            if (hasSide(4)) {
                insertSide({0, 0, 0}, {nx, ny, ngz}, 4, outputVolume);
            }

            if (hasSide(5)) {
                insertSide({0, 0, nz - ngz}, {nx, ny, nz}, 5, outputVolume);
            }
        }
    }

}


void CudaCartesianCellExchanger::makeStreams(const volume::Volume&
    inputVolume) {
    memoryStreams.resize(inputVolume.getNumberOfVariables());

    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        memoryStreams[var].resize(6);
        int dimensions = inputVolume.getDimensions();

        for (int side = 0; side < 2 * dimensions; ++side) {
            CUDA_SAFE_CALL(hipStreamCreate(&memoryStreams[var][side]));
        }

        for (int corner = 0; corner < numberOfCorners; ++corner) {
            CUDA_SAFE_CALL(hipStreamCreate(&memoryStreams[var][corner]));
        }
    }
}

void CudaCartesianCellExchanger::makeBuffers(const volume::Volume&
    inputVolume,
    const volume::Volume& outputVolume) {

    makeBuffersSides(inputVolume, outputVolume);
    makeBuffersCorners(inputVolume, outputVolume);
}

void CudaCartesianCellExchanger::makeBuffersSides(const volume::Volume& inputVolume,
                                                    const volume::Volume& outputVolume) {

    buffers.resize(inputVolume.getNumberOfVariables());
    cpuBuffersSend.resize(buffers.size());
    cpuBuffersReceive.resize(buffers.size());

    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        buffers[var].resize(6);
        cpuBuffersSend[var].resize(6);
        cpuBuffersReceive[var].resize(6);

        for (int side = 0; side < 6; ++side) {
            if (hasSide(side)) {
                const int nx = (side > 1) * inputVolume.getTotalNumberOfXCells() +
                    (side < 2) * inputVolume.getNumberOfXGhostCells();

                const int ny = (side != 2) * (side != 3) * inputVolume.getTotalNumberOfYCells()
                    +
                    ((side == 2) + (side == 3)) * inputVolume.getNumberOfYGhostCells();


                const int nz = (side != 4) * (side != 5) * inputVolume.getTotalNumberOfZCells()
                    +
                    ((side == 4) + (side == 5)) * inputVolume.getNumberOfZGhostCells();



                buffers[var][side] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);

                cpuBuffersSend[var][side].resize(nx * ny * nz, 0);
                cpuBuffersReceive[var][side].resize(nx * ny * nz, 0);
                //alsfvm::make_shared<alsfvm::memory::HostMemory<real>>(nx, ny, nz);
            }
        }
    }
}

void CudaCartesianCellExchanger::makeBuffersCorners(const volume::Volume& inputVolume,
                                                    const volume::Volume& outputVolume) {

    buffersCorners.resize(inputVolume.getNumberOfVariables());
    cpuBuffersSendCorners.resize(buffers.size());
    cpuBuffersReceiveCorners.resize(buffers.size());

    const auto ghostCells = outputVolume.getNumberOfGhostCells();
    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        buffersCorners[var].resize(6);
        cpuBuffersSendCorners[var].resize(6);
        cpuBuffersReceiveCorners[var].resize(6);

        for (int corner = 0; corner < numberOfCorners; ++corner) {
            if (hasCorner(corner)) {
                const auto position = cartesian::getCoordinates(corner, ivec3{2,2,2});



                const int nx = ghostCells.x;
                const int ny = std::max(ghostCells.y, 1);
                const int nz = std::max(ghostCells.z, 1);



                buffersCorners[var][corner] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);

                cpuBuffersSendCorners[var][corner].resize(nx * ny * nz, 0);
                cpuBuffersReceiveCorners[var][corner].resize(nx * ny * nz, 0);
                //alsfvm::make_shared<alsfvm::memory::HostMemory<real>>(nx, ny, nz);
            }
        }
    }
}



}
}
