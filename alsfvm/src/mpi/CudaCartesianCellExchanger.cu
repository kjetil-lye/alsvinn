#include "hip/hip_runtime.h"
/* Copyright (c) 2018 ETH Zurich, Kjetil Olsen Lye
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "alsfvm/mpi/CudaCartesianCellExchanger.hpp"
#include "alsutils/mpi/safe_call.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"
namespace alsfvm {
namespace mpi {

namespace {

__global__ void extractSideDevice(memory::View<real> output,
    memory::View<const real> input,
    ivec3 start, ivec3 end) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const int nx = end.x - start.x;
    const int ny = end.y - start.y;
    const int nz = end.z - start.z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int inputX = x + start.x;
    const int inputY = y + start.y;
    const int inputZ = z + start.z;

    output.at(x, y, z) = input.at(inputX, inputY, inputZ);

}


__global__ void insertSideDevice(memory::View<real> output,
    memory::View< real> input,
    ivec3 start, ivec3 end) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const int nx = end.x - start.x;
    const int ny = end.y - start.y;
    const int nz = end.z - start.z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int outputX = x + start.x;
    const int outputY = y + start.y;
    const int outputZ = z + start.z;

    output.at(outputX, outputY, outputZ) = input.at(x, y, z);

}

}
CudaCartesianCellExchanger::CudaCartesianCellExchanger(ConfigurationPtr&
    configuration, const ivec6& neighbours, const std::array<int, 8>& cornerNeighbours)
    : configuration(configuration), neighbours(neighbours), cornerNeighbours(cornerNeighbours) {


}

RequestContainer CudaCartesianCellExchanger::exchangeCells(
    volume::Volume& outputVolume,
    const volume::Volume& inputVolume) {

    const int dimensions = inputVolume.getDimensions();

    if (buffers.size() == 0) {
        makeBuffers(inputVolume);
        makeStreams(inputVolume);

        receiveRequests.resize(inputVolume.getNumberOfVariables());
        sendRequests.resize(inputVolume.getNumberOfVariables());

        for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
            receiveRequests[var].resize(2 * dimensions);
            sendRequests[var].resize(2 * dimensions);
        }
    }


    extractSides(inputVolume);

    auto oppositeSide = [&](int side) {
        const int i = side % 2;
        return (i + 1) % 2 + (side / 2) * 2;
    };

    RequestContainer container;


    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {

        for (int side = 0; side < 2 * dimensions; ++side) {
            if (hasSide(side)) {
                CUDA_SAFE_CALL(hipStreamSynchronize(memoryStreams[var][side]));
                sendRequests[var][side] = (Request::isend(cpuBuffersSend[var][side],
                            cpuBuffersSend[var][side].size(),
                            MPI_DOUBLE, neighbours[side],
                            var * 6 + side,
                            *configuration));
            }

            if (hasSide(oppositeSide(side))) {
                receiveRequests[var][oppositeSide(side)] = Request::ireceive(
                        cpuBuffersReceive[var][oppositeSide(side)],
                        cpuBuffersReceive[var][oppositeSide(side)].size(),
                        MPI_DOUBLE, neighbours[oppositeSide(side)],
                        var * 6 + side,
                        *configuration);
            }
        }
    }


    insertSides(outputVolume);

    CUDA_SAFE_CALL(hipDeviceSynchronize());

    RequestContainer emptyContainer;
    return emptyContainer;
}

int CudaCartesianCellExchanger::getNumberOfActiveSides() const {
    int activeSides = 0;

    for (int i = 0; i < 6; ++i) {
        if (hasSide(i)) {
            activeSides++;
        }
    }

    return activeSides;
}

real CudaCartesianCellExchanger::max(real number) {
    real max;
    MPI_SAFE_CALL(MPI_Allreduce(&number, &max, 1, MPI_DOUBLE, MPI_MAX,
            configuration->getCommunicator()));
    return max;
}
bool CudaCartesianCellExchanger::hasSide(int side) const {
    return neighbours[side] != -1;
}

void CudaCartesianCellExchanger::extractSide(const ivec3& start,
    const ivec3& end,
    int side,
    const volume::Volume& inputVolume) {
    for (int var  = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        const auto diff = end - start;
        const int size = diff.x * diff.y * diff.z;

        if (size <= 0) {
            return;
        }

        const int numberOfThreads = 512;

        extractSideDevice <<< (size + numberOfThreads - 1) / numberOfThreads,
                          numberOfThreads,
                          0, memoryStreams[var][side] >>> (buffers[var][side]->getView(),
                              inputVolume.getScalarMemoryArea(var)->getView(),
                              start,
                              end);


        CUDA_SAFE_CALL(hipMemcpyAsync(cpuBuffersSend[var][side].data(),
                buffers[var][side]->getPointer(),
                buffers[var][side]->getSize()*sizeof(real),
                hipMemcpyDeviceToHost,
                memoryStreams[var][side]));


    }


}

ivec6 CudaCartesianCellExchanger::getNeighbours() const {
    return neighbours;
}
void CudaCartesianCellExchanger::extractSides(const volume::Volume&
    inputVolume) {
    const int nx = inputVolume.getTotalNumberOfXCells();
    const int ny = inputVolume.getTotalNumberOfYCells();
    const int nz = inputVolume.getTotalNumberOfZCells();

    const int ngx = inputVolume.getNumberOfXGhostCells();
    const int ngy = inputVolume.getNumberOfYGhostCells();
    const int ngz = inputVolume.getNumberOfZGhostCells();

    const int dimensions = inputVolume.getDimensions();

    if (hasSide(0)) {
        extractSide({ngx, 0, 0}, {2 * ngx, ny, nz}, 0, inputVolume);
    }

    if (hasSide(1)) {

        extractSide({nx - 2 * ngx, 0, 0}, {nx - ngx, ny, nz}, 1, inputVolume);
    }

    if (dimensions > 1) {

        if (hasSide(2)) {

            extractSide({0, ngy, 0}, {nx, 2 * ngy, nz}, 2, inputVolume);
        }

        if (hasSide(3)) {

            extractSide({0, ny - 2 * ngy, 0}, {nx, ny - ngy, nz}, 3, inputVolume);
        }

        if (dimensions > 2) {
            if (hasSide(4)) {

                extractSide({0, 0, ngz}, {nx, ny, 2 * ngz}, 4, inputVolume);
            }

            if (hasSide(5)) {

                extractSide({0, 0, nz - 2 * ngz}, {nx, ny, nz - ngz}, 5, inputVolume);
            }
        }

    }

}


void CudaCartesianCellExchanger::insertSide(const ivec3& start,
    const ivec3& end,
    int side,
    volume::Volume& outputVolume) {
    for (int var  = 0; var < outputVolume.getNumberOfVariables(); ++var) {


        const auto diff = end - start;
        const int size = diff.x * diff.y * diff.z;

        if (size == 0) {
            return;
        }


        //sendRequests[var][side]->wait();
        receiveRequests[var][side]->wait();
        CUDA_SAFE_CALL(hipMemcpyAsync(buffers[var][side]->getPointer(),
                cpuBuffersReceive[var][side].data(),
                buffers[var][side]->getSize()*sizeof(real),
                hipMemcpyHostToDevice,
                memoryStreams[var][side]));

        const int numberOfThreads = 512;
        insertSideDevice <<< (size + numberOfThreads - 1) / numberOfThreads,
                         numberOfThreads,
                         0, memoryStreams[var][side] >>> (
                             outputVolume.getScalarMemoryArea(var)->getView(),
                             buffers[var][side]->getView(),
                             start,
                             end);

    }
}


void CudaCartesianCellExchanger::insertSides( volume::Volume& outputVolume) {
    const int nx = outputVolume.getTotalNumberOfXCells();
    const int ny = outputVolume.getTotalNumberOfYCells();
    const int nz = outputVolume.getTotalNumberOfZCells();

    const int ngx = outputVolume.getNumberOfXGhostCells();
    const int ngy = outputVolume.getNumberOfYGhostCells();
    const int ngz = outputVolume.getNumberOfZGhostCells();


    const int dimensions = outputVolume.getDimensions();

    if (hasSide(0)) {
        insertSide({0, 0, 0}, {ngx, ny, nz}, 0, outputVolume);
    }

    if (hasSide(1)) {
        insertSide({nx - ngx, 0, 0}, {nx, ny, nz}, 1, outputVolume);
    }

    if (dimensions > 1) {

        if (hasSide(2)) {
            insertSide({0, 0, 0}, {nx, ngy, nz}, 2, outputVolume);
        }

        if (hasSide(3)) {
            insertSide({0, ny - ngy, 0}, {nx, ny, nz}, 3, outputVolume);
        }

        if (dimensions > 2 ) {
            if (hasSide(4)) {
                insertSide({0, 0, 0}, {nx, ny, ngz}, 4, outputVolume);
            }

            if (hasSide(5)) {
                insertSide({0, 0, nz - ngz}, {nx, ny, nz}, 5, outputVolume);
            }
        }
    }

}


void CudaCartesianCellExchanger::makeStreams(const volume::Volume&
    inputVolume) {
    memoryStreams.resize(inputVolume.getNumberOfVariables());

    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        memoryStreams[var].resize(6);
        int dimensions = inputVolume.getDimensions();

        for (int side = 0; side < 2 * dimensions; ++side) {
            CUDA_SAFE_CALL(hipStreamCreate(&memoryStreams[var][side]));
            //memoryStreams[var][side]=0;
        }
    }
}

void CudaCartesianCellExchanger::makeBuffers(const volume::Volume&
    inputVolume) {
    buffers.resize(inputVolume.getNumberOfVariables());
    cpuBuffersSend.resize(buffers.size());
    cpuBuffersReceive.resize(buffers.size());

    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        buffers[var].resize(6);
        cpuBuffersSend[var].resize(6);
        cpuBuffersReceive[var].resize(6);

        for (int side = 0; side < 6; ++side) {
            if (hasSide(side)) {
                const int nx = (side > 1) * inputVolume.getTotalNumberOfXCells() +
                    (side < 2) * inputVolume.getNumberOfXGhostCells();

                const int ny = (side != 2) * (side != 3) * inputVolume.getTotalNumberOfYCells()
                    +
                    ((side == 2) + (side == 3)) * inputVolume.getNumberOfYGhostCells();


                const int nz = (side != 4) * (side != 5) * inputVolume.getTotalNumberOfZCells()
                    +
                    ((side == 4) + (side == 5)) * inputVolume.getNumberOfZGhostCells();



                buffers[var][side] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);

                cpuBuffersSend[var][side].resize(nx * ny * nz, 0);
                cpuBuffersReceive[var][side].resize(nx * ny * nz, 0);
                //alsfvm::make_shared<alsfvm::memory::HostMemory<real>>(nx, ny, nz);
            }
        }
    }
}
}
}
