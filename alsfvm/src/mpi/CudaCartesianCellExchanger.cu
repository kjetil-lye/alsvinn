#include "hip/hip_runtime.h"
/* Copyright (c) 2018 ETH Zurich, Kjetil Olsen Lye
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 * 
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 * 
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "alsfvm/mpi/CudaCartesianCellExchanger.hpp"
#include "alsutils/mpi/safe_call.hpp"
#include "alsfvm/cuda/cuda_utils.hpp"
#include "alsutils/mpi/mpi_types.hpp"
#include "alsfvm/gpu_array.hpp"
#define L std::cout << __LINE__ << __FILE__ << std::endl;
namespace alsfvm {
namespace mpi {

namespace {

template<int numberOfSides, int numberOfVariables>
__global__ void extractSideDevice(
    gpu_array<gpu_array<memory::View<real>, numberOfSides>, numberOfVariables> output,
    gpu_array<memory::View<const real>, numberOfVariables> input,
    gpu_array<ivec3, numberOfSides> starts,
    gpu_array<ivec3, numberOfSides> ends,
    gpu_array<bool, numberOfSides> activeSides) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;

    const int side = blockIdx.y;

    if (!activeSides[side]) {
        return;
    }
    const int var = blockIdx.z;

    const int nx = ends[side].x - starts[side].x;
    const int ny = ends[side].y - starts[side].y;
    const int nz = ends[side].z - starts[side].z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int inputX = x + starts[side].x;
    const int inputY = y + starts[side].y;
    const int inputZ = z + starts[side].z;

    output[var][side].at(x, y, z) = input[var].at(inputX, inputY, inputZ);

}


template<int numberOfSides, int numberOfVariables>
__global__ void insertSideDevice(gpu_array<memory::View<real>, numberOfVariables> output,
                                 gpu_array<gpu_array<memory::View<const real>, numberOfSides>, numberOfVariables> input,
                                 gpu_array<ivec3, numberOfSides> starts,
                                 gpu_array<ivec3, numberOfSides> ends,
                                 gpu_array<bool, numberOfSides> activeSides) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;


    const int side = blockIdx.y;


    if (!activeSides[side]) {
        return;
    }
    const int var = blockIdx.z;

    const int nx = ends[side].x - starts[side].x;
    const int ny = ends[side].y - starts[side].y;
    const int nz = ends[side].z - starts[side].z;

    const int x = index % nx;
    const int y = (index / nx) % ny;
    const int z = (index / nx) / ny;

    if (x >= nx || y >= ny || z >= nz) {
        return;
    }

    const int outputX = x + starts[side].x;
    const int outputY = y + starts[side].y;
    const int outputZ = z + starts[side].z;

    output[var].at(outputX, outputY, outputZ) = input[var][side].at(x, y, z);
}

}
CudaCartesianCellExchanger::CudaCartesianCellExchanger(ConfigurationPtr&
    configuration, const ivec6& neighbours)
    : configuration(configuration), neighbours(neighbours) {


}

RequestContainer CudaCartesianCellExchanger::exchangeCells(
    volume::Volume& outputVolume,
    const volume::Volume& inputVolume) {

    const int dimensions = inputVolume.getDimensions();
#ifdef ALSVINN_MPI_GPU_DIRECT
    if (buffersSend.size() == 0) {
#else
    if (buffers.size() == 0) {
#endif
        makeBuffers(inputVolume);
        makeStreams(inputVolume);

        receiveRequests.resize(inputVolume.getNumberOfVariables());
        sendRequests.resize(inputVolume.getNumberOfVariables());

        for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
            receiveRequests[var].resize(2 * dimensions);
            sendRequests[var].resize(2 * dimensions);
        }
    }

L
    callExtractSides(inputVolume);
L
    auto oppositeSide = [&](int s) {
        int d = s / 2;
        int i = s % 2;

        return (i + 1) % 2 + d * 2;
    };
L
    RequestContainer container;

L
    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
L
        for (int side = 0; side < 2 * dimensions; ++side) {
            if (hasSide(side)) {
                CUDA_SAFE_CALL(hipStreamSynchronize(memoryStreams[var][side]));
#ifndef ALSVINN_MPI_GPU_DIRECT
                sendRequests[var][side] = (Request::isend(cpuBuffersSend[var][side],
                            cpuBuffersSend[var][side].size(),
                            alsutils::mpi::MpiTypes<real>::MPI_Real, neighbours[side],
                            var * 6 + side,
                            *configuration));
#else
                sendRequests[var][side] = (Request::isend(*buffersSend[var][side],
                            buffersSend[var][side]->getSize(),
                            alsutils::mpi::MpiTypes<real>::MPI_Real, neighbours[side],
                            var * 6 + side,
                            *configuration));
#endif
            }
L
            if (hasSide(oppositeSide(side))) {
    L
#ifndef ALSVINN_MPI_GPU_DIRECT
                receiveRequests[var][oppositeSide(side)] = Request::ireceive(
                        cpuBuffersReceive[var][oppositeSide(side)],
                        cpuBuffersReceive[var][oppositeSide(side)].size(),
                        alsutils::mpi::MpiTypes<real>::MPI_Real, neighbours[oppositeSide(side)],
                        var * 6 + side,
                        *configuration);
#else
                receiveRequests[var][oppositeSide(side)] = Request::ireceive(
                        *buffersReceive[var][oppositeSide(side)],
                        buffersReceive[var][oppositeSide(side)]->getSize(),
                        alsutils::mpi::MpiTypes<real>::MPI_Real, neighbours[oppositeSide(side)],
                        var * 6 + side,
                        *configuration);
#endif
    L
            }
        }
    }
L

    callInsertSides(outputVolume);
L
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    RequestContainer emptyContainer;
    return emptyContainer;
}

int CudaCartesianCellExchanger::getNumberOfActiveSides() const {
    int activeSides = 0;

    for (int i = 0; i < 6; ++i) {
        if (hasSide(i)) {
            activeSides++;
        }
    }

    return activeSides;
}

real CudaCartesianCellExchanger::max(real number) {
    real max;
    MPI_SAFE_CALL(MPI_Allreduce(&number, &max, 1, alsutils::mpi::MpiTypes<real>::MPI_Real, MPI_MAX,
            configuration->getCommunicator()));
    return max;
}
bool CudaCartesianCellExchanger::hasSide(int side) const {
    return neighbours[side] != -1;
}


template<int numberOfSides, int numberOfVariables>
void CudaCartesianCellExchanger::extractSide(const gpu_array<ivec3, numberOfSides>& start,
    const gpu_array<ivec3, numberOfSides>& end,
    const volume::Volume& inputVolume,
     gpu_array<bool, numberOfSides> activeSides) {



    gpu_array<gpu_array<memory::View<real>, numberOfSides>, numberOfVariables> output;
    gpu_array<memory::View<const real>, numberOfVariables> input;

L


    std::array<int, numberOfSides> sizes;
    for (int side = 0; side < numberOfSides; ++side) {

        const auto diff = end[side] - start[side];
        const int size = diff.x * diff.y * diff.z;

        sizes[side] = size;


    }
L
    // sanity check
    for(int side = 1; side < numberOfSides; ++side) {
        if (sizes[side] != sizes[side-1]) {
            THROW("We need every side to have the same number of elements for the exchange.");
        }
    }

    if (sizes[0] == 0) {
        return;
    }

L
    for (int var  = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        input[var] = inputVolume.getScalarMemoryArea(var)->getView();
        for (int side = 0; side < numberOfSides; ++side) {
            if (hasSide(side)) {
#ifndef ASVINN_MPI_GPU_DIRECT
                output[var][side] = buffers[var][side]->getView();
#else
                output[var][side] = buffersSend[var][side]->getView();
#endif
            }
        }
    }


L

    const int numberOfThreads = 512;


    dim3 gridDim;
    gridDim.x = (sizes[0] + numberOfThreads - 1) / numberOfThreads;
    gridDim.y = numberOfSides;
    gridDim.z = numberOfVariables;

    extractSideDevice<numberOfSides, numberOfVariables> <<<gridDim,
                      numberOfThreads,
                      0, memoryStreams[0][0] >>> (output,
                      input,
                      start,
                      end, activeSides);
L
#ifndef ALSVINN_MPI_GPU_DIRECT
    for (int var  = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        for (int side = 0; side < numberOfSides; ++side) {
            if (hasSide(side)) {
                CUDA_SAFE_CALL(hipMemcpyAsync(cpuBuffersSend[var][side].data(),
                   buffers[var][side]->getPointer(),
                   buffers[var][side]->getSize()*sizeof(real),
                   hipMemcpyDeviceToHost,
                   memoryStreams[0][0]));
            }
        }
    }
#endif

L


}

ivec6 CudaCartesianCellExchanger::getNeighbours() const {
    return neighbours;
}

void CudaCartesianCellExchanger::callExtractSides(const volume::Volume&
    inputVolume) {


    const int dimensions = inputVolume.getDimensions();

    switch(dimensions) {
    case 1:
        extractSides<2>(inputVolume);
        break;
    case 2:
        extractSides<4>(inputVolume);
        break;
    case 3:
        extractSides<6>(inputVolume);
        break;
    default:
        THROW("Unexpected dimension " << dimensions);
    }


}

template<int numberOfSides>
void CudaCartesianCellExchanger::callExtractSide(const gpu_array<ivec3, numberOfSides>& start,
                                                 const gpu_array<ivec3, numberOfSides>& end,
                                                 const volume::Volume& inputvolume,
                                                 gpu_array<bool, numberOfSides> activeSides) {

    const auto numberOfVariables = inputvolume.getNumberOfVariables();
    switch(numberOfVariables) {
    case 1:
        extractSide<numberOfSides, 1>(start, end, inputvolume, activeSides);
        break;
    case 2:
        extractSide<numberOfSides, 2>(start, end, inputvolume, activeSides);
        break;
    case 3:
        extractSide<numberOfSides, 3>(start, end, inputvolume, activeSides);
        break;
    case 4:
        extractSide<numberOfSides, 4>(start, end, inputvolume, activeSides);
        break;
    case 5:
        extractSide<numberOfSides, 5>(start, end, inputvolume, activeSides);
        break;
    case 6:
        extractSide<numberOfSides, 6>(start, end, inputvolume, activeSides);
        break;
    default:
        THROW("Unexpected number of variables " << numberOfVariables);
    }


}

template<int numberOfSides>
void CudaCartesianCellExchanger::extractSides(const volume::Volume&
    inputVolume) {
    const int nx = inputVolume.getTotalNumberOfXCells();
    const int ny = inputVolume.getTotalNumberOfYCells();
    const int nz = inputVolume.getTotalNumberOfZCells();

    const int ngx = inputVolume.getNumberOfXGhostCells();
    const int ngy = inputVolume.getNumberOfYGhostCells();
    const int ngz = inputVolume.getNumberOfZGhostCells();


    const int dimensions = inputVolume.getDimensions();

    gpu_array<ivec3, numberOfSides> starts;
    gpu_array<ivec3, numberOfSides> ends;
    gpu_array<bool, numberOfSides> activeSides;

    for (size_t side = 0; side < numberOfSides; ++side) {
        activeSides[side] = hasSide(side);
    }

    starts[0] = {ngx, 0, 0};
    ends[0] = {2 * ngx, ny, nz};


    starts[1] = {nx-2*ngx, 0, 0};
    ends[1] = {nx-ngx, ny, nz};


    if (dimensions > 1) {


        starts[2] = {0, ngy, 0};
        ends[2] = {nx, 2 * ngy, nz};

        starts[3] = {0, ny - 2* ngy, 0};
        ends[3] = {nx, ny-ngy, nz};


        if (dimensions > 2) {

            starts[4] = {0, 0, ngz};
            ends[4] = {nx, ny, 2 * ngz};



            starts[5] = {0, 0, nz - 2 * ngz};
            ends[5] = {nx, ny, nz - ngz};

        }

    }

    callExtractSide<numberOfSides>(starts, ends, inputVolume, activeSides);

}


void CudaCartesianCellExchanger::callInsertSides(volume::Volume&
    outputVolume) {


    const int dimensions = outputVolume.getDimensions();

    switch(dimensions) {
    case 1:
        insertSides<2>(outputVolume);
        break;
    case 2:
        insertSides<4>(outputVolume);
        break;
    case 3:
        insertSides<6>(outputVolume);
        break;
    default:
        THROW("Unexpected dimension " << dimensions);
    }


}

template<int numberOfSides>
void CudaCartesianCellExchanger::callInsertSide(const gpu_array<ivec3, numberOfSides>& start,
                                                 const gpu_array<ivec3, numberOfSides>& end,
                                                 volume::Volume& outputVolume,
                                                 gpu_array<bool, numberOfSides> activeSides) {

    const auto numberOfVariables = outputVolume.getNumberOfVariables();
    switch(numberOfVariables) {
    case 1:
        insertSide<numberOfSides, 1>(start, end, outputVolume, activeSides);
        break;
    case 2:
        insertSide<numberOfSides, 2>(start, end, outputVolume, activeSides);
        break;
    case 3:
        insertSide<numberOfSides, 3>(start, end, outputVolume, activeSides);
        break;
    case 4:
        insertSide<numberOfSides, 4>(start, end, outputVolume, activeSides);
        break;
    case 5:
        insertSide<numberOfSides, 5>(start, end, outputVolume, activeSides);
        break;
    case 6:
        insertSide<numberOfSides, 6>(start, end, outputVolume, activeSides);
        break;
    default:
        THROW("Unexpected number of variables " << numberOfVariables);
    }


}


template<int numberOfSides, int numberOfVariables>
void CudaCartesianCellExchanger::insertSide(const gpu_array<ivec3, numberOfSides>& start,
                                            const gpu_array<ivec3, numberOfSides>& end,
                                            volume::Volume& outputVolume,
                                            gpu_array<bool, numberOfSides> activeSides) {


    gpu_array<gpu_array<memory::View<const real>, numberOfSides>, numberOfVariables> input;
    gpu_array<memory::View<real>, numberOfVariables> output;

L


    std::array<int, numberOfSides> sizes;
    for (int side = 0; side < numberOfSides; ++side) {

        const auto diff = end[side] - start[side];
        const int size = diff.x * diff.y * diff.z;

        sizes[side] = size;


    }
L
    // sanity check
    for(int side = 1; side < numberOfSides; ++side) {
        if (sizes[side] != sizes[side-1]) {
            THROW("We need every side to have the same number of elements for the exchange.");
        }
    }

    if (sizes[0] == 0) {
        return;
    }
L

    for (int var  = 0; var < outputVolume.getNumberOfVariables(); ++var) {
        output[var] = outputVolume.getScalarMemoryArea(var)->getView();
        for (int side = 0; side < numberOfSides; ++side) {

            if (hasSide(side)) {
#ifndef ALSVINN_MPI_GPU_DIRECT
                input[var][side] = buffers[var][side]->getConstView();
#else
                input[var][side] = buffersReceive[var][side]->getConstView();
#endif
            }

        }
    }
L

#ifndef ALSVINN_MPI_GPU_DIRECT

    for (int var  = 0; var < outputVolume.getNumberOfVariables(); ++var) {
        for (int side = 0; side < numberOfSides; ++side) {
            if (hasSide(side)) {
                sendRequests[var][side]->wait();
                receiveRequests[var][side]->wait();

                CUDA_SAFE_CALL(hipMemcpyAsync(buffers[var][side]->getPointer(),
                    cpuBuffersReceive[var][side].data(),
                    buffers[var][side]->getSize()*sizeof(real),
                    hipMemcpyHostToDevice,
                    memoryStreams[0][0]));
            }
        }
    }
#else
    for (int var  = 0; var < outputVolume.getNumberOfVariables(); ++var) {
        for (int side = 0; side < numberOfSides; ++side) {
            if (hasSide(side)) {
                sendRequests[var][side]->wait();
                receiveRequests[var][side]->wait();
            }
        }
    }
#endif
L


    const int numberOfThreads = 512;

L
    dim3 gridDim;
    gridDim.x = (sizes[0] + numberOfThreads - 1) / numberOfThreads;
    gridDim.y = numberOfSides;
    gridDim.z = numberOfVariables;

    insertSideDevice<numberOfSides, numberOfVariables> <<< gridDim,
                         numberOfThreads,
                         0, memoryStreams[0][0] >>> (
                             output,
                             input,
                             start,
                             end, activeSides);

L
}

template<int numberOfSides>
void CudaCartesianCellExchanger::insertSides( volume::Volume& outputVolume) {
    const int nx = outputVolume.getTotalNumberOfXCells();
    const int ny = outputVolume.getTotalNumberOfYCells();
    const int nz = outputVolume.getTotalNumberOfZCells();

    const int ngx = outputVolume.getNumberOfXGhostCells();
    const int ngy = outputVolume.getNumberOfYGhostCells();
    const int ngz = outputVolume.getNumberOfZGhostCells();


    const int dimensions = outputVolume.getDimensions();

    gpu_array<ivec3, numberOfSides> starts;
    gpu_array<ivec3, numberOfSides> ends;
    gpu_array<bool, numberOfSides> activeSides;

    for (size_t side = 0; side < numberOfSides; ++side) {
        activeSides[side] = hasSide(side);
    }


    starts[0] = {0, 0, 0};
    ends[0] =  {ngx, ny, nz};

    starts[1] = {nx - ngx, 0, 0};
    ends[1] = {nx, ny, nz};

    if (dimensions > 1) {

       starts[2] = {0, 0, 0};
       ends[2] = {nx, ngy, nz};

       starts[3] = {0, ny - ngy, 0};
       ends[3] = {nx, ny, nz};
       if (dimensions > 2 ) {
            starts[4] = {0, 0, 0};
            ends[4] = {nx, ny, ngz};

            starts[5] = {0, 0, nz - ngz};
            ends[5] = {nx, ny, nz};
        }
    }

    callInsertSide<numberOfSides>(starts, ends, outputVolume, activeSides);

}


void CudaCartesianCellExchanger::makeStreams(const volume::Volume&
    inputVolume) {
    memoryStreams.resize(inputVolume.getNumberOfVariables());

    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
        memoryStreams[var].resize(6);
        int dimensions = inputVolume.getDimensions();

        for (int side = 0; side < 2 * dimensions; ++side) {
            CUDA_SAFE_CALL(hipStreamCreate(&memoryStreams[var][side]));
            //memoryStreams[var][side]=0;
        }
    }
}

void CudaCartesianCellExchanger::makeBuffers(const volume::Volume&
    inputVolume) {
#ifndef ALSVINN_MPI_GPU_DIRECT
    buffers.resize(inputVolume.getNumberOfVariables());
    cpuBuffersSend.resize(buffers.size());
    cpuBuffersReceive.resize(buffers.size());
#else
    buffersSend.resize(inputVolume.getNumberOfVariables());
    buffersReceive.resize(buffersSend.size());
#endif
    for (int var = 0; var < inputVolume.getNumberOfVariables(); ++var) {
#ifndef ALSVINN_MPI_GPU_DIRECT
        buffers[var].resize(6);
        cpuBuffersSend[var].resize(6);
        cpuBuffersReceive[var].resize(6);
#else
        buffersSend[var].resize(6);
        buffersReceive[var].resize(6);
#endif

        for (int side = 0; side < 6; ++side) {
            if (hasSide(side)) {
                const int nx = (side > 1) * inputVolume.getTotalNumberOfXCells() +
                    (side < 2) * inputVolume.getNumberOfXGhostCells();

                const int ny = (side != 2) * (side != 3) * inputVolume.getTotalNumberOfYCells()
                    +
                    ((side == 2) + (side == 3)) * inputVolume.getNumberOfYGhostCells();


                const int nz = (side != 4) * (side != 5) * inputVolume.getTotalNumberOfZCells()
                    +
                    ((side == 4) + (side == 5)) * inputVolume.getNumberOfZGhostCells();
#ifndef ALSVINN_MPI_GPU_DIRECT
                buffers[var][side] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);

                cpuBuffersSend[var][side].resize(nx * ny * nz, 0);
                cpuBuffersReceive[var][side].resize(nx * ny * nz, 0);
#else
                buffersSend[var][side] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);

                buffersReceive[var][side] = alsfvm::make_shared<alsfvm::cuda::CudaMemory<real>>(nx, ny,
                        nz);
#endif
                //alsfvm::make_shared<alsfvm::memory::HostMemory<real>>(nx, ny, nz);
            }
        }
    }
}
}
}
